

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define MASK_N 2
#define MASK_X 5
#define MASK_Y 5
#define SCALE  8

unsigned char *image_s = NULL;     // source image array
unsigned char *image_t = NULL;     // target image array
FILE *fp_s = NULL;                 // source file handler
FILE *fp_t = NULL;                 // target file handler

unsigned int   width, height;      // image width, image height
unsigned int   rgb_raw_data_offset;// RGB raw data offset
unsigned char  bit_per_pixel;      // bit per pixel
unsigned short byte_per_pixel;     // byte per pixel

// bitmap header
unsigned char header[54] = {
		0x42,        // identity : B
		0x4d,        // identity : M
		0, 0, 0, 0,  // file size
		0, 0,        // reserved1
		0, 0,        // reserved2
		54, 0, 0, 0, // RGB data offset
		40, 0, 0, 0, // struct BITMAPINFOHEADER size
		0, 0, 0, 0,  // bmp width
		0, 0, 0, 0,  // bmp height
		1, 0,        // planes
		24, 0,       // bit per pixel
		0, 0, 0, 0,  // compression
		0, 0, 0, 0,  // data size
		0, 0, 0, 0,  // h resolution
		0, 0, 0, 0,  // v resolution
		0, 0, 0, 0,  // used colors
		0, 0, 0, 0   // important colors
};

// sobel mask (5x5 version)
int
		mask[MASK_N][MASK_X][MASK_Y] = {
		{{ -1, -4, -6, -4, -1},
				{ -2, -8,-12, -8, -2},
				{  0,  0,  0,  0,  0},
				{  2,  8, 12,  8,  2},
				{  1,  4,  6,  4,  1}}
		,
		{{ -1, -2,  0,  2,  1},
				{ -4, -8,  0,  8,  4},
				{ -6,-12,  0, 12,  6},
				{ -4, -8,  0,  8,  4},
				{ -1, -2,  0,  2,  1}}
};


int
read_bmp (const char *fname_s) {
	fp_s = fopen(fname_s, "rb");
	if (fp_s == NULL) {
		printf("fopen fp_s error\n");
		return -1;
	}

	// move offset to 10 to find rgb raw data offset
	fseek(fp_s, 10, SEEK_SET);
	fread(&rgb_raw_data_offset, sizeof(unsigned int), 1, fp_s);

	// move offset to 18 to get width & height;
	fseek(fp_s, 18, SEEK_SET);
	fread(&width,  sizeof(unsigned int), 1, fp_s);
	fread(&height, sizeof(unsigned int), 1, fp_s);

	// get bit per pixel
	fseek(fp_s, 28, SEEK_SET);
	fread(&bit_per_pixel, sizeof(unsigned short), 1, fp_s);
	byte_per_pixel = bit_per_pixel / 8;

	// move offset to rgb_raw_data_offset to get RGB raw data
	fseek(fp_s, rgb_raw_data_offset, SEEK_SET);

	// image_s = (unsigned char *) malloc((size_t)width * height * byte_per_pixel);
	hipHostMalloc( (void**)&image_s, (size_t)width * height * byte_per_pixel, hipHostMallocDefault);

	fread(image_s, sizeof(unsigned char), (size_t)(long) width * height * byte_per_pixel, fp_s);

	return 0;
}

__global__ void
sobel_Kernel (
		unsigned char* cuda_image_t,
		const unsigned char* cuda_image_s,
		int* global_cuda_mask,
		unsigned int* cuda_width,
		unsigned int *cuda_height,
		short *cuda_byte_per_pixel
) {
	int  x, y, i, v, u;            // for loop counter
	int  R, G, B;                  // color of R, G, B
	double val[MASK_N*3] = {0.0};
	int adjustX, adjustY, xBound, yBound;
	unsigned int width = *cuda_width;
	unsigned int height = *cuda_height;
	short byte_per_pixel = *cuda_byte_per_pixel;


	__shared__ int share_cuda_mask[MASK_N][MASK_X][MASK_Y];
	// int id = (threadIdx.z * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
	if(threadIdx.x<MASK_N*MASK_X*MASK_Y)
	{
		int dim_1, dim_2, dim_3;
		dim_1 = threadIdx.x % MASK_Y;
		dim_2 = threadIdx.x / MASK_Y;
		dim_3 = dim_2 / MASK_X;
		dim_2 = dim_2 % MASK_X;
		share_cuda_mask[dim_3][dim_2][dim_1] = global_cuda_mask[threadIdx.x];
	}
	__syncthreads();


	y = threadIdx.x+blockIdx.x*blockDim.x;
	if(y<height)
		for (x = 0; x < width; ++x) {

			for (i = 0; i < MASK_N; ++i) {
				adjustX = (MASK_X % 2) ? 1 : 0;
				adjustY = (MASK_Y % 2) ? 1 : 0;
				xBound = MASK_X /2;
				yBound = MASK_Y /2;

				val[i*3+2] = 0.0;
				val[i*3+1] = 0.0;
				val[i*3] = 0.0;

				for (v = -yBound; v < yBound + adjustY; ++v) {
					for (u = -xBound; u < xBound + adjustX; ++u) {
						if ((x + u) >= 0 && (x + u) < width && y + v >= 0 && y + v < height) {
							R = cuda_image_s[byte_per_pixel * (width * (y+v) + (x+u)) + 2];
							G = cuda_image_s[byte_per_pixel * (width * (y+v) + (x+u)) + 1];
							B = cuda_image_s[byte_per_pixel * (width * (y+v) + (x+u)) + 0];
							val[i*3+2] += R * share_cuda_mask[i][u + xBound][v + yBound];
							val[i*3+1] += G * share_cuda_mask[i][u + xBound][v + yBound];
							val[i*3+0] += B * share_cuda_mask[i][u + xBound][v + yBound];
						}
					}
				}

			}

			double totalR = 0.0;
			double totalG = 0.0;
			double totalB = 0.0;
			for (i = 0; i < MASK_N; ++i) {
				totalR += val[i*3+2] * val[i*3+2];
				totalG += val[i*3+1] * val[i*3+1];
				totalB += val[i*3+0] * val[i*3+0];
			}

			totalR = sqrt(totalR) / SCALE;
			totalG = sqrt(totalG) / SCALE;
			totalB = sqrt(totalB) / SCALE;
			const unsigned char cR = (totalR > 255.0) ? 255 : totalR;
			const unsigned char cG = (totalG > 255.0) ? 255 : totalG;
			const unsigned char cB = (totalB > 255.0) ? 255 : totalB;
			cuda_image_t[ byte_per_pixel * (width * y + x) + 2 ] = cR;
			cuda_image_t[ byte_per_pixel * (width * y + x) + 1 ] = cG;
			cuda_image_t[ byte_per_pixel * (width * y + x) + 0 ] = cB;
		}

}

int
write_bmp (const char *fname_t) {
	unsigned int file_size; // file size

	fp_t = fopen(fname_t, "wb");
	if (fp_t == NULL) {
		printf("fopen fname_t error\n");
		return -1;
	}

	// file size
	file_size = width * height * byte_per_pixel + rgb_raw_data_offset;
	header[2] = (unsigned char)(file_size & 0x000000ff);
	header[3] = (file_size >> 8)  & 0x000000ff;
	header[4] = (file_size >> 16) & 0x000000ff;
	header[5] = (file_size >> 24) & 0x000000ff;

	// width
	header[18] = width & 0x000000ff;
	header[19] = (width >> 8)  & 0x000000ff;
	header[20] = (width >> 16) & 0x000000ff;
	header[21] = (width >> 24) & 0x000000ff;

	// height
	header[22] = height &0x000000ff;
	header[23] = (height >> 8)  & 0x000000ff;
	header[24] = (height >> 16) & 0x000000ff;
	header[25] = (height >> 24) & 0x000000ff;

	// bit per pixel
	header[28] = bit_per_pixel;

	// write header
	fwrite(header, sizeof(unsigned char), rgb_raw_data_offset, fp_t);

	// write image
	fwrite(image_t, sizeof(unsigned char), (size_t)(long)width * height * byte_per_pixel, fp_t);

	fclose(fp_s);
	fclose(fp_t);

	return 0;
}

int
init_device ()
{
	hipSetDevice(0);
	return 0;
}

int
main(int argc, char **argv) {
	init_device();

	const char *input = "candy.bmp";
	if (argc > 1) input = argv[1];
	read_bmp(input); // 24 bit gray level image

	unsigned char * cuda_image_t;
	unsigned char * cuda_image_s;
	int  * global_cuda_mask ;
	unsigned int * cuda_width;
	unsigned int * cuda_height;
	short * cuda_byte_per_pixel;


	size_t mask_size = sizeof(int)*MASK_N*MASK_X*MASK_Y;
	size_t image_size = (size_t)width * height * byte_per_pixel;

	hipMalloc((void**)&cuda_image_s, image_size);
	hipMalloc((void**)&cuda_image_t, image_size);
	hipMalloc((void**)&global_cuda_mask, mask_size);
	hipMalloc((void**)&cuda_width, 1*sizeof(unsigned int));
	hipMalloc((void**)&cuda_height, 1*sizeof(unsigned int));
	hipMalloc((void**)&cuda_byte_per_pixel, 1*sizeof(short));

	hipMemcpy(cuda_image_s, image_s, image_size, hipMemcpyHostToDevice);
	// free (image_s);
	// cudaFreeHost(image_s);
	hipMemcpy(global_cuda_mask,(int*)mask, mask_size, hipMemcpyHostToDevice);
	hipMemcpy(cuda_width, &width, 1*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(cuda_height, &height, 1*sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(cuda_byte_per_pixel, &byte_per_pixel, 1*sizeof(short), hipMemcpyHostToDevice);



	int thread_number = 64;
	sobel_Kernel<<<height/thread_number+1, thread_number>>>(cuda_image_t, cuda_image_s, global_cuda_mask, cuda_width, cuda_height, cuda_byte_per_pixel);

	// image_t = (unsigned char *) malloc((size_t) width * height * byte_per_pixel);
	hipHostMalloc( (void**)&image_t, image_size, hipHostMallocDefault);

	hipMemcpy(image_t, cuda_image_t, image_size, hipMemcpyDeviceToHost);

	write_bmp("result.bmp");

	// free (image_t);
	// cudaFreeHost(image_t);
	// cudaFree(cuda_image_t);
	// cudaFree(cuda_image_s);
	// cudaFree(global_cuda_mask);
	// cudaFree(cuda_width);
	// cudaFree(cuda_width);
	// cudaFree(cuda_byte_per_pixel);

}
